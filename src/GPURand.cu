#include "hip/hip_runtime.h"
#include "GPURand.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <set>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

using namespace Random;

std::vector<unsigned int>  RandomHelper::getStaticAssemblyBuffer(unsigned int min, unsigned int max) {

	std::vector<unsigned int> result;

	//unsigned int max = 0xffff;
	unsigned int ui = min;
	while (ui <= max) {
		result.push_back(ui);
		ui++;
	}


	return result;
}

std::vector<std::vector<unsigned int>> RandomHelper::getRandomizers(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width) {
	return getRndBuffer(seed, min, max, len, width, false);
}

std::vector<std::vector<unsigned int>> RandomHelper::getRndBuffer(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width, bool singlePool) {

	std::vector<std::vector<unsigned int>> results;

	std::vector<uint64_t> seedMods = RandomHelper::randRange(width, min, max);

	for (int64_t i = 0; i < width; i++) {
		if (!singlePool || i == 0) {
			//time for the devrnd seed

			thrust::default_random_engine rng(seed * seedMods[i]);

			thrust::uniform_int_distribution<unsigned int> dist(min, max);

			//thrust::host_vector<unsigned int> h_vec(buffer_len);
			std::vector<unsigned int> result(len);

			thrust::generate(result.begin(), result.end(), [&] { return dist(rng); });

			// Transfer data to the device.
			thrust::device_vector<unsigned int> d_vec = result; //h_vec;

			// Transfer data back to host.
			thrust::copy(d_vec.begin(), d_vec.end(), result.begin());

			results.push_back(result);
		}
		else {
			//make a copy instead of generate
			std::vector<unsigned int> result = results[0];
			results.push_back(result);
		}
	}

	return results;

}

std::vector<std::vector<unsigned int>> RandomHelper::getRndBuffer(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width, bool singlePool, bool allowDupes) {
	std::vector<std::vector<unsigned int>> results = getRndBuffer(seed, min, max, len, width, singlePool);

	if (!allowDupes) {
		for (int i = 0; i < results.size(); i++) {
			std::vector<unsigned int> draftResults = results[i];
			std::set<unsigned int> dedupe;
			for (int j = 0; j < draftResults.size(); j++) {
				dedupe.insert(draftResults[j]);
			}
			std::vector<unsigned int> finalResults;
			std::copy(dedupe.begin(), dedupe.end(), std::inserter(finalResults, finalResults.end()));
			results[i] = finalResults;
		}
	}
	return results;
}

std::vector<uint64_t> RandomHelper::randRange(uint64_t buffer_len, uint64_t min, uint64_t max) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<uint64_t> dist(min, max);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<uint64_t> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<uint64_t> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}
