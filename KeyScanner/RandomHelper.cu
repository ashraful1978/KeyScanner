#include "hip/hip_runtime.h"
#include "RandomHelper.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <set>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>




int RandomHelper::randTest()
{
	int MAX_VALUE = 64535;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(1337);
	thrust::uniform_int_distribution<int> dist;
	thrust::host_vector<int> h_vec(32 << 20);
	thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<int> d_vec = h_vec;

	// Sort data on the device.
	//thrust::sort(d_vec.begin(), d_vec.end());

	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	std::vector<int> results;
	for (int i : h_vec) {
		if (i <= MAX_VALUE) results.push_back(i);
	}

	
	return  EXIT_SUCCESS;

}

std::vector<unsigned int> RandomHelper::getStaticAssemblyBuffer() {

	std::vector<unsigned int> result;

	unsigned int max = 0xffff;
	unsigned int ui = 0;
	while (ui <= max) {
		result.push_back(ui);
		ui++;
	}


	return result;
}

std::vector<std::vector<unsigned int>> RandomHelper::getRandomizers(uint64_t seed, uint64_t len, uint64_t width) {
	return getRndBuffer(seed, 0, 0xffff, len, width, false);
}

std::vector<std::vector<unsigned int>> RandomHelper::getRndBuffer(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width, bool singlePool) {

	std::vector<std::vector<unsigned int>> results;

	for (int64_t i = 0; i < width; i++) {
		if (!singlePool || i == 0) {
			//time for the devrnd seed

			thrust::default_random_engine rng(seed * i);

			thrust::uniform_int_distribution<unsigned int> dist(min, max);

			//thrust::host_vector<unsigned int> h_vec(buffer_len);
			std::vector<unsigned int> result(len);

			thrust::generate(result.begin(), result.end(), [&] { return dist(rng); });

			// Transfer data to the device.
			thrust::device_vector<unsigned int> d_vec = result; //h_vec;

			// Transfer data back to host.
			thrust::copy(d_vec.begin(), d_vec.end(), result.begin());

			results.push_back(result);
		}
		else {
			//make a copy instead of generate
			std::vector<unsigned int> result = results[0];
			results.push_back(result);
		}
	}

	return results;

}

std::vector<std::vector<unsigned int>> RandomHelper::getRndBuffer(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width, bool singlePool, bool allowDupes) {
	std::vector<std::vector<unsigned int>> results = getRndBuffer(seed, min, max, len, width, singlePool);
	
	if (!allowDupes) {
		for (int i = 0; i < results.size(); i++) {
			std::vector<unsigned int> draftResults = results[i];
			std::set<unsigned int> dedupe;
			for (int j = 0; j < draftResults.size(); j++) {
				dedupe.insert(draftResults[j]);
			}
			std::vector<unsigned int> finalResults;
			std::copy(dedupe.begin(), dedupe.end(), std::inserter(finalResults, finalResults.end()));
			results[i] = finalResults;
		}
	}
	return results;
}

std::vector<unsigned int> RandomHelper::getRndBuffer16(uint64_t buffer_len) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<unsigned int> dist(0, 0xffff);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<unsigned int> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<unsigned int> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}
std::vector<unsigned int> RandomHelper::getRndBuffer32(uint64_t buffer_len) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<unsigned int> dist(0, 0xffffffff);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<unsigned int> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<unsigned int> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}

std::vector<uint64_t> RandomHelper::randRange(uint64_t buffer_len, uint64_t min, uint64_t max) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<uint64_t> dist(min, max);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<uint64_t> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<uint64_t> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}


std::vector<uint64_t> RandomHelper::sequentialRange(uint64_t buffer_len, uint64_t min, uint64_t max, bool reversed) {

	std::vector<uint64_t> results;
	uint64_t count = 0;
	uint64_t running_value = reversed ? max : min;


		while (count < buffer_len) {
			results.push_back(running_value);
			count++;
			//conditionally iterate
			if (reversed) {
				running_value--;
				if (running_value == min) running_value = max;
			}
			else {
				running_value++;
				if (running_value = max) running_value = min;
			}
		}

	return results;
}

std::vector<uint64_t> RandomHelper::getRndRange(uint64_t seed, uint64_t min_value, uint64_t max_value, uint64_t size) {
	std::vector<uint64_t> results;

	while (results.size() <= size) {
		thrust::host_vector<uint64_t> rndBuffer = getRndBatch(seed, min_value, max_value, size);
		for (uint64_t i : rndBuffer) {
			//printf("%d \n", i );
			if (i <= max_value) results.push_back(i);
			if (results.size() == size) break;;
		}
	}
	return results;
}

std::vector<uint64_t> RandomHelper::getRndRange(uint64_t seed, std::vector<uint64_t> values, uint64_t size) {
	std::vector<uint64_t> results;

	while (results.size() <= size) {
		thrust::host_vector<uint64_t> rndBuffer = getRndBatch(seed, 0, values.size()-1, size);
		for (uint64_t i : rndBuffer) {
			uint64_t value = values[i];
			//printf("%d \n", i );
			results.push_back(value);
			if (results.size() == size) break;;
		}
	}
	return results;
}

thrust::host_vector<uint64_t> RandomHelper::getRndBatch(uint64_t seed, uint64_t min_value, uint64_t max_value, uint64_t size) {
	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(seed);
	thrust::uniform_int_distribution<uint64_t> dist(min_value, max_value);
	//thrust::host_vector<int> h_vec(32 << 20);
	thrust::host_vector<uint64_t> h_vec(size);
	thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<uint64_t> d_vec = h_vec;

	// Sort data on the device.  - sort freaks out, I think b/c of sort namespace colisions 
	//thrust::sort(d_vec.begin(), d_vec.end());

	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	dist.reset();

	return h_vec;
}


secp256k1::uint256 RandomHelper::getDefaultRandomRange(secp256k1::uint256 min, secp256k1::uint256 max)
{
	secp256k1::uint256 result;
	secp256k1::uint256 range = max.sub(min);

	unsigned char targetByteSize = (range.getBitRange() + 31) / 32;

	for (int i = 0; i < 8; i++) {
		if (targetByteSize > i) {
			result.v[i] = 0; //CudaRng::rndDefault.getChunk();
			if (targetByteSize > i && targetByteSize <= i + 1 && range.v[i] != 0 && result.v[i] > range.v[i]) {
				result.v[i] %= range.v[i];
			}
		}
	}

	return result.add(min);
}

std::vector<uint64_t> RandomHelper::getCPURndBatch(uint64_t seed, uint64_t min_value, uint64_t max_value, uint64_t size) {
	// Generate Random using CPU and Standard Engine
	std::random_device rd;
	std::default_random_engine* eng = new std::default_random_engine(rd());
	eng->seed(seed);

	std::uniform_int_distribution<uint64_t>* dist = new std::uniform_int_distribution<uint64_t>(min_value, max_value);

	std::vector<uint64_t> h_vec;//(size);
	
	uint64_t ct = 0;

	while (ct <= size) {
		uint64_t value = (*dist)(*eng);
		h_vec.push_back(value);
		ct++;
	}
	
	delete eng;
	delete dist;

	return h_vec;
}

std::vector<uint64_t> RandomHelper::getCPURndRange(uint64_t seed, std::vector<uint64_t> values, uint64_t size) {
	std::vector<uint64_t> results;

	while (results.size() <= size) {
		std::vector<uint64_t> rndBuffer = getCPURndBatch(seed, 0, values.size() - 1, size);
		for (uint64_t i : rndBuffer) {
			uint64_t value = values[i];
			//printf("%d \n", i );
			results.push_back(value);
			if (results.size() == size) break;;
		}
	}
	return results;
}


std::vector<secp256k1::uint256> RandomHelper::sortKeys(std::vector<secp256k1::uint256> keys) {
	std::vector<secp256k1::uint256> results;
	thrust::host_vector<secp256k1::uint256> hostKeys;
	//thrust::device_vector<secp256k1::uint256> deviceKeys = keys;

	for (int k = 0; k < keys.size(); k++) {
		hostKeys.push_back(keys[k]);
	}

	//thrust::copy(keys.begin(), keys.end(), hostKeys.begin());
	thrust::sort(hostKeys.begin(), hostKeys.end());
	//thrust::sort(deviceKeys.begin(), deviceKeys.end());

	// Transfer data back to host.
	//thrust::copy(deviceKeys.begin(), deviceKeys.end(), results.begin());
	//thrust::copy(hostKeys.begin(), hostKeys.end(), results.begin());

	for (int k = 0; k < hostKeys.size(); k++) {
		results.push_back(hostKeys[k]);
	}

	return results;
}


std::vector<secp256k1::uint256> RandomHelper::getDistances(std::vector<secp256k1::uint256> keys) {
	std::vector<secp256k1::uint256> results;
	thrust::host_vector<secp256k1::uint256> hostKeys;

	//iterate through the keys, determine the intra-key distances
	secp256k1::uint256 lastKey;
	for (int k = 0; k < keys.size(); k++) {
		if (k > 0) {
			results.push_back(keys[k] - lastKey);
		}
		else {
			results.push_back(0);
		}
		lastKey = keys[k];
	}

	return results;
}

