#include "hip/hip_runtime.h"
#include "RandomHelper.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <set>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>


using namespace Random;

std::vector<unsigned int> RandomHelper::getStaticAssemblyBuffer(unsigned int min, unsigned int max) {

	std::vector<unsigned int> result;

	//unsigned int max = 0xffff;
	unsigned int ui = min;
	while (ui <= max) {
		result.push_back(ui);
		ui++;
	}


	return result;
}

std::vector<std::vector<unsigned int>> RandomHelper::getRandomizers(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width) {
	return getRndBuffer(seed, min, max, len, width, false);
}

std::vector<std::vector<unsigned int>> RandomHelper::getRndBuffer(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width, bool singlePool) {

	std::vector<std::vector<unsigned int>> results;

	std::vector<uint64_t> seedMods = RandomHelper::randRange(width, 0, 69000000);
	for (int64_t i = 0; i < width; i++) {
		if (!singlePool || i == 0) {
			//time for the devrnd seed

			thrust::default_random_engine rng(seed * seedMods[i]);

			thrust::uniform_int_distribution<unsigned int> dist(min, max);

			//thrust::host_vector<unsigned int> h_vec(buffer_len);
			std::vector<unsigned int> result(len);

			thrust::generate(result.begin(), result.end(), [&] { return dist(rng); });

			// Transfer data to the device.
			thrust::device_vector<unsigned int> d_vec = result; //h_vec;

			// Transfer data back to host.
			thrust::copy(d_vec.begin(), d_vec.end(), result.begin());

			results.push_back(result);
		}
		else {
			//make a copy instead of generate
			std::vector<unsigned int> result = results[0];
			results.push_back(result);
		}
	}

	return results;

}

std::vector<std::vector<unsigned int>> RandomHelper::getRndBuffer(uint64_t seed, unsigned int min, unsigned int max, uint64_t len, uint64_t width, bool singlePool, bool allowDupes) {
	std::vector<std::vector<unsigned int>> results = getRndBuffer(seed, min, max, len, width, singlePool);
	
	if (!allowDupes) {
		for (int i = 0; i < results.size(); i++) {
			std::vector<unsigned int> draftResults = results[i];
			std::set<unsigned int> dedupe;
			for (int j = 0; j < draftResults.size(); j++) {
				dedupe.insert(draftResults[j]);
			}
			std::vector<unsigned int> finalResults;
			std::copy(dedupe.begin(), dedupe.end(), std::inserter(finalResults, finalResults.end()));
			results[i] = finalResults;
		}
	}
	return results;
}

std::vector<unsigned int> RandomHelper::getRndBuffer16(uint64_t buffer_len) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<unsigned int> dist(0, 0xffff);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<unsigned int> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<unsigned int> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}
std::vector<unsigned int> RandomHelper::getRndBuffer32(uint64_t buffer_len) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<unsigned int> dist(0, 0xffffffff);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<unsigned int> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<unsigned int> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}

std::vector<uint64_t> RandomHelper::randRange(uint64_t buffer_len, uint64_t min, uint64_t max) {
	//time for the devrnd seed
	time_t now = time(nullptr);
	time_t mnow = now * 1000;

	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(mnow);

	thrust::uniform_int_distribution<uint64_t> dist(min, max);

	//thrust::host_vector<unsigned int> h_vec(buffer_len);
	std::vector<uint64_t> results(buffer_len);

	thrust::generate(results.begin(), results.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<uint64_t> d_vec = results; //h_vec;


	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), results.begin());

	return results;
}


std::vector<uint64_t> RandomHelper::sequentialRange(uint64_t buffer_len, uint64_t min, uint64_t max, bool reversed) {

	std::vector<uint64_t> results;
	uint64_t count = 0;
	uint64_t running_value = reversed ? max : min;


		while (count < buffer_len) {
			results.push_back(running_value);
			count++;
			//conditionally iterate
			if (reversed) {
				running_value--;
				if (running_value == min) running_value = max;
			}
			else {
				running_value++;
				if (running_value = max) running_value = min;
			}
		}

	return results;
}

std::vector<uint64_t> RandomHelper::getRndRange(uint64_t seed, uint64_t min_value, uint64_t max_value, uint64_t size) {
	std::vector<uint64_t> results;

	while (results.size() < size) {
		thrust::host_vector<uint64_t> rndBuffer = getRndBatch(seed, min_value, max_value, size);
		for (uint64_t i : rndBuffer) {
			//printf("%d \n", i );
			if (i <= max_value) results.push_back(i);
			if (results.size() == size) break;
		}
		if (results.size() == size) break;
	}
	return results;
}

std::vector<uint64_t> RandomHelper::getRndRange(uint64_t seed, std::vector<uint64_t> values, uint64_t size) {
	std::vector<uint64_t> results;

	while (results.size() <= size) {
		thrust::host_vector<uint64_t> rndBuffer = getRndBatch(seed, 0, values.size()-1, size);
		for (uint64_t i : rndBuffer) {
			uint64_t value = values[i];
			//printf("%d \n", i );
			results.push_back(value);
			if (results.size() == size) break;;
		}
	}
	return results;
}

thrust::host_vector<uint64_t> RandomHelper::getRndBatch(uint64_t seed, uint64_t min_value, uint64_t max_value, uint64_t size) {
	// Generate 32M random numbers serially.
	thrust::default_random_engine rng(seed);
	thrust::uniform_int_distribution<uint64_t> dist(min_value, max_value);
	//thrust::host_vector<int> h_vec(32 << 20);
	thrust::host_vector<uint64_t> h_vec(size);
	thrust::generate(h_vec.begin(), h_vec.end(), [&] { return dist(rng); });

	// Transfer data to the device.
	thrust::device_vector<uint64_t> d_vec = h_vec;

	// Sort data on the device.  - sort freaks out, I think b/c of sort namespace colisions 
	//thrust::sort(d_vec.begin(), d_vec.end());

	// Transfer data back to host.
	thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

	dist.reset();

	return h_vec;
}


secp256k1::uint256 RandomHelper::getDefaultRandomRange(secp256k1::uint256 min, secp256k1::uint256 max)
{
	secp256k1::uint256 result;
	secp256k1::uint256 range = max.sub(min);

	unsigned char targetByteSize = (range.getBitRange() + 31) / 32;

	for (int i = 0; i < 8; i++) {
		if (targetByteSize > i) {
			result.v[i] = 0; //CudaRng::rndDefault.getChunk();
			if (targetByteSize > i && targetByteSize <= i + 1 && range.v[i] != 0 && result.v[i] > range.v[i]) {
				result.v[i] %= range.v[i];
			}
		}
	}

	return result.add(min);
}

std::vector<uint64_t> RandomHelper::getCPURndBatch(uint64_t seed, uint64_t min_value, uint64_t max_value, uint64_t size) {
	// Generate Random using CPU and Standard Engine
	std::random_device rd;
	std::default_random_engine* eng = new std::default_random_engine(rd());
	eng->seed(seed);

	std::uniform_int_distribution<uint64_t>* dist = new std::uniform_int_distribution<uint64_t>(min_value, max_value);

	std::vector<uint64_t> h_vec;//(size);
	
	uint64_t ct = 0;

	while (ct <= size) {
		uint64_t value = (*dist)(*eng);
		h_vec.push_back(value);
		ct++;
	}
	
	delete eng;
	delete dist;

	return h_vec;
}

std::vector<uint64_t> RandomHelper::getCPURndRange(uint64_t seed, std::vector<uint64_t> values, uint64_t size) {
	std::vector<uint64_t> results;

	while (results.size() <= size) {
		std::vector<uint64_t> rndBuffer = getCPURndBatch(seed, 0, values.size() - 1, size);
		for (uint64_t i : rndBuffer) {
			uint64_t value = values[i];
			//printf("%d \n", i );
			results.push_back(value);
			if (results.size() == size) break;;
		}
	}
	return results;
}


std::vector<secp256k1::uint256> RandomHelper::sortKeys(std::vector<secp256k1::uint256> keys) {
	std::vector<secp256k1::uint256> results;
	thrust::host_vector<secp256k1::uint256> hostKeys;
	//thrust::device_vector<secp256k1::uint256> deviceKeys = keys;

	for (int k = 0; k < keys.size(); k++) {
		hostKeys.push_back(keys[k]);
	}

	//thrust::copy(keys.begin(), keys.end(), hostKeys.begin());
	thrust::sort(hostKeys.begin(), hostKeys.end());
	//thrust::sort(deviceKeys.begin(), deviceKeys.end());

	// Transfer data back to host.
	//thrust::copy(deviceKeys.begin(), deviceKeys.end(), results.begin());
	//thrust::copy(hostKeys.begin(), hostKeys.end(), results.begin());

	for (int k = 0; k < hostKeys.size(); k++) {
		results.push_back(hostKeys[k]);
	}

	return results;
}


std::vector<secp256k1::uint256> RandomHelper::getDistances(std::vector<secp256k1::uint256> keys, uint64_t truncate) {
	std::vector<secp256k1::uint256> results;

	//iterate through the keys, determine the intra-key distances
	//uint32_t scaler = std::pow(16, truncate);

	//secp256k1::uint256 scaler256 = secp256k1::uint256("4096");
	secp256k1::uint256 lastKey;
	secp256k1::uint256 thisKey;
	secp256k1::uint256 thisDistance;
	secp256k1::uint256 referenceDistance;
	for (int k = 0; k < keys.size(); k++) {

		//Logger::log(LogLevel::Debug, "ThisK1: " + _startingKeys.Keys[k].toString());
		thisKey = keys[k];
		//thisKey = keys[k].div(scaler);
		//thisKey = keys[k].rShift(scaler);

		//thisKey = keys[k].div(;

		if (k > 0) {
			//truncate the end of the comparison key - so we are measuruing random-only differences not including a right-padded sequential
			thisDistance = thisKey.sub(lastKey);

			//pad the right of the distance so that it can be cleanly added without disturbing the sequentials
			//thisDistance = thisDistance.mul(scaler);

			if (thisDistance == 0) {
				thisDistance = referenceDistance;
			}
			else {
				referenceDistance = thisDistance;
			}

			results.push_back(thisDistance);
		}
		else {
			results.push_back(0);
		}
		lastKey = thisKey;
	}

	return results;
}

secp256k1::uint256 RandomHelper::getDistanceAverage(std::vector<secp256k1::uint256> keys) {
	secp256k1::uint256 result;

	secp256k1::uint256 thisDistance;
	secp256k1::uint256 distanceSum;
	secp256k1::uint256 keyCount = secp256k1::uint256(0);
	secp256k1::uint256 one = secp256k1::uint256(1);

	for (int k = 0; k < keys.size(); k++) {

		thisDistance = keys[k];
		distanceSum = distanceSum.add(thisDistance);
		keyCount = keyCount.add(one);
	}

	result = distanceSum.div(keyCount);
	return result;
}

secp256k1::uint256 RandomHelper::getDistanceMean(std::vector<secp256k1::uint256> keys) {
	secp256k1::uint256 result;

	uint64_t startKey = keys.size() * 0.25;
	uint64_t endKey = keys.size() - startKey;


	secp256k1::uint256 thisDistance;
	secp256k1::uint256 distanceSum;
	secp256k1::uint256 keyCount = secp256k1::uint256(0);
	secp256k1::uint256 one = secp256k1::uint256(1);

	for (int k = startKey; k < endKey; k++) {

		thisDistance = keys[k];
		distanceSum = distanceSum.add(thisDistance);
		keyCount = keyCount.add(one);
	}

	result = distanceSum.div(keyCount);
	return result;
}


